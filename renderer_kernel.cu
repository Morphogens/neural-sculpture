#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include <ATen/ATen.h>

#define PI 3.14159265358979323846f
namespace {
    
    __device__ __forceinline__ float DegToRad(const float &deg) { return (deg * (PI / 180.f)); }

    
    __device__ __forceinline__ float length(
                            const float x, 
                            const float y, 
                            const float z) {
        return sqrtf(powf(x, 2) + powf(y, 2) + powf(z, 2));
    }

    // Cross product
    __device__ __forceinline__ float cross_x(
                            const float a_x, 
                            const float a_y, 
                            const float a_z, 
                            const float b_x, 
                            const float b_y, 
                            const float b_z) {
        return a_y * b_z - a_z * b_y;
    }

    
    __device__ __forceinline__ float cross_y(
                            const float a_x, 
                            const float a_y, 
                            const float a_z, 
                            const float b_x, 
                            const float b_y, 
                            const float b_z) {
        return a_z * b_x - a_x * b_z;
    }

    
    __device__ __forceinline__ float cross_z(
                            const float a_x, 
                            const float a_y, 
                            const float a_z, 
                            const float b_x, 
                            const float b_y, 
                            const float b_z) {
        return a_x * b_y - a_y * b_x;
    }

    __global__ void GenerateRay(
                float* origins, 
                float* directions,
                float* origin_image_distances, 
                float* pixel_distances,
                const int width, 
                const int height, 
                const float eye_x, 
                const float eye_y, 
                const float eye_z) {

        const float at_x = 0;
        const float at_y = 0;
        const float at_z = 0;
        const float up_x = 0;
        const float up_y = 1;
        const float up_z = 0;

        // Compute camera view volume
        const float top = tan(DegToRad(30));
        const float bottom = -top;
        const float right = (__int2float_rd(width) / __int2float_rd(height)) * top;
        const float left = -right;
        
        // Compute local base
        const float w_x = (eye_x - at_x) / length(eye_x - at_x, eye_y - at_y, eye_z - at_z);
        const float w_y = (eye_y - at_y) / length(eye_x - at_x, eye_y - at_y, eye_z - at_z);
        const float w_z = (eye_z - at_z) / length(eye_x - at_x, eye_y - at_y, eye_z - at_z);
        const float cross_up_w_x = cross_x(up_x, up_y, up_z, w_x, w_y, w_z);
        const float cross_up_w_y = cross_y(up_x, up_y, up_z, w_x, w_y, w_z);
        const float cross_up_w_z = cross_z(up_x, up_y, up_z, w_x, w_y, w_z);
        const float u_x = (cross_up_w_x) / length(cross_up_w_x, cross_up_w_y, cross_up_w_z);
        const float u_y = (cross_up_w_y) / length(cross_up_w_x, cross_up_w_y, cross_up_w_z);
        const float u_z = (cross_up_w_z) / length(cross_up_w_x, cross_up_w_y, cross_up_w_z);
        const float v_x = cross_x(w_x, w_y, w_z, u_x, u_y, u_z);
        const float v_y = cross_y(w_x, w_y, w_z, u_x, u_y, u_z);
        const float v_z = cross_z(w_x, w_y, w_z, u_x, u_y, u_z);


        const int pixel_index = blockIdx.x * blockDim.x + threadIdx.x;

        if (pixel_index < width * height) {
            const int x = pixel_index % width;
            const int y = pixel_index / width;
            const int i = 3 * pixel_index;

            // Compute point on view plane
            // Ray passes through the center of the pixel
            const float view_plane_x = left + (right - left) * (__int2float_rd(x) + 0.5) / __int2float_rd(width);
            const float view_plane_y = top - (top - bottom) * (__int2float_rd(y) + 0.5) / __int2float_rd(height);
            const float s_x = view_plane_x * u_x + view_plane_y * v_x - w_x;
            const float s_y = view_plane_x * u_y + view_plane_y * v_y - w_y;
            const float s_z = view_plane_x * u_z + view_plane_y * v_z - w_z;
            origins[i] = eye_x;
            origins[i+1] = eye_y;
            origins[i+2] = eye_z;

            
            directions[i] = s_x / length(s_x, s_y, s_z);
            directions[i+1] = s_y / length(s_x, s_y, s_z);
            directions[i+2] = s_z / length(s_x, s_y, s_z);

            origin_image_distances[pixel_index] = length(s_x, s_y, s_z);
            pixel_distances[pixel_index] = (right - left) / __int2float_rd(width);
            
        }
    }  

    // Check if a point is inside
    __device__ __forceinline__ bool InsideBoundingBox(
                                        const float p_x, 
                                        const float p_y, 
                                        const float p_z,
                                        const float bounding_box_min_x,
                                        const float bounding_box_min_y,
                                        const float bounding_box_min_z,
                                        const float bounding_box_max_x,
                                        const float bounding_box_max_y,
                                        const float bounding_box_max_z) {

        return (p_x >= bounding_box_min_x) && (p_x <= bounding_box_max_x) &&
               (p_y >= bounding_box_min_y) && (p_y <= bounding_box_max_y) &&
               (p_z >= bounding_box_min_z) && (p_z <= bounding_box_max_z);
    }

    // Compute the distance along the ray between the point and the bounding box  
    __device__ float Distance(
        const float reached_point_x,
        const float reached_point_y,
        const float reached_point_z,
        float direction_x,
        float direction_y,
        float direction_z,
        const float bounding_box_min_x,
        const float bounding_box_min_y,
        const float bounding_box_min_z,
        const float bounding_box_max_x,
        const float bounding_box_max_y,
        const float bounding_box_max_z) {

        float dist = -1.f;
        direction_x = direction_x / length(direction_x, direction_y, direction_z);
        direction_y = direction_y / length(direction_x, direction_y, direction_z);
        direction_z = direction_z / length(direction_x, direction_y, direction_z);

        // For each axis count any excess distance outside box extents
        float v = reached_point_x;
        float d = direction_x;
        if (dist == -1) {
            if ((v < bounding_box_min_x) && (d > 0)) { dist = (bounding_box_min_x - v) / d; }
            if ((v > bounding_box_max_x) && (d < 0)) { dist = (bounding_box_max_x - v) / d; }
        } else {
            if ((v < bounding_box_min_x) && (d > 0)) { dist = fmaxf(dist, (bounding_box_min_x - v) / d); }
            if ((v > bounding_box_max_x) && (d < 0)) { dist = fmaxf(dist, (bounding_box_max_x - v) / d); }
        }

        v = reached_point_y;
        d = direction_y;
        if (dist == -1) {
            if ((v < bounding_box_min_y) && (d > 0)) { dist = (bounding_box_min_y - v) / d; }
            if ((v > bounding_box_max_y) && (d < 0)) { dist = (bounding_box_max_y - v) / d; }
        } else {
            if ((v < bounding_box_min_y) && (d > 0)) { dist = fmaxf(dist, (bounding_box_min_y - v) / d); }
            if ((v > bounding_box_max_y) && (d < 0)) { dist = fmaxf(dist, (bounding_box_max_y - v) / d); }
        }

        v = reached_point_z;
        d = direction_z;
        if (dist == -1) {
            if ((v < bounding_box_min_z) && (d > 0)) { dist = (bounding_box_min_z - v) / d; }
            if ((v > bounding_box_max_z) && (d < 0)) { dist = (bounding_box_max_z - v) / d; }
        } else {
            if ((v < bounding_box_min_z) && (d > 0)) { dist = fmaxf(dist, (bounding_box_min_z - v) / d); }
            if ((v > bounding_box_max_z) && (d < 0)) { dist = fmaxf(dist, (bounding_box_max_z - v) / d); }
        }

        return dist;
    }

    __device__ __forceinline__ int flat(float const x, float const y, float const z,
                                        int const grid_res_x, int const grid_res_y, int const grid_res_z) {
        return __int2float_rd(z) + __int2float_rd(y) * grid_res_z + __int2float_rd(x) * grid_res_z * grid_res_y;
    }

    // Get the signed distance value at the specific point
    __device__ float ValueAt(
        const float* grid,
        const float reached_point_x,
        const float reached_point_y,
        const float reached_point_z, 
        const float direction_x,
        const float direction_y,
        const float direction_z,
        const float bounding_box_min_x,
        const float bounding_box_min_y,
        const float bounding_box_min_z,
        const float bounding_box_max_x,
        const float bounding_box_max_y,
        const float bounding_box_max_z,
        const int grid_res_x, 
        const int grid_res_y,
        const int grid_res_z,
        const bool first_time) {
        
        // Check if we are outside the BBOX
        if (!InsideBoundingBox(reached_point_x, reached_point_y, reached_point_z, 
                               bounding_box_min_x,
                               bounding_box_min_y,
                               bounding_box_min_z,
                               bounding_box_max_x,
                               bounding_box_max_y,
                               bounding_box_max_z)) {

            // If it is the first time, then the ray has not entered the grid
            if (first_time) {
                 
                return Distance(reached_point_x, reached_point_y, reached_point_z,
                                direction_x, direction_y, direction_z,
                                bounding_box_min_x,
                                bounding_box_min_y,
                                bounding_box_min_z,
                                bounding_box_max_x,
                                bounding_box_max_y,
                                bounding_box_max_z) + 0.00001f;
            }

            // Otherwise, the ray has left the grid
            else {
                return -1;
            }
        }
        
        // Compute voxel size
        float voxel_size = (bounding_box_max_x - bounding_box_min_x) / (grid_res_x - 1);

        // Compute the the minimum point of the intersecting voxel
        float min_index_x = floorf((reached_point_x - bounding_box_min_x) / voxel_size);
        float min_index_y = floorf((reached_point_y - bounding_box_min_y) / voxel_size);
        float min_index_z = floorf((reached_point_z - bounding_box_min_z) / voxel_size);

        // Check whether the ray intersects the vertex with the last index of the axis
        // If so, we should record the previous index
        if (min_index_x == (bounding_box_max_x - bounding_box_min_x) / voxel_size) {
            min_index_x = (bounding_box_max_x - bounding_box_min_x) / voxel_size - 1;
        }
        if (min_index_y == (bounding_box_max_y - bounding_box_min_y) / voxel_size) {
            min_index_y = (bounding_box_max_y - bounding_box_min_y) / voxel_size - 1;
        }
        if (min_index_z == (bounding_box_max_z - bounding_box_min_z) / voxel_size) {
            min_index_z = (bounding_box_max_z - bounding_box_min_z) / voxel_size - 1;
        }

        // Linear interpolate along x axis the eight values
        const float tx = (reached_point_x - (bounding_box_min_x + min_index_x * voxel_size)) / voxel_size;
        const float c01 = (1.f - tx) * grid[flat(min_index_x, min_index_y, min_index_z, grid_res_x, grid_res_y, grid_res_z)]
         + tx * grid[flat(min_index_x+1, min_index_y, min_index_z, grid_res_x, grid_res_y, grid_res_z)];
        const float c23 = (1.f - tx) * grid[flat(min_index_x, min_index_y+1, min_index_z, grid_res_x, grid_res_y, grid_res_z)]
         + tx * grid[flat(min_index_x+1, min_index_y+1, min_index_z, grid_res_x, grid_res_y, grid_res_z)];
        const float c45 = (1.f - tx) * grid[flat(min_index_x, min_index_y, min_index_z+1, grid_res_x, grid_res_y, grid_res_z)]
         + tx * grid[flat(min_index_x+1, min_index_y, min_index_z+1, grid_res_x, grid_res_y, grid_res_z)];
        const float c67 = (1.f - tx) * grid[flat(min_index_x, min_index_y+1, min_index_z+1, grid_res_x, grid_res_y, grid_res_z)]
         + tx * grid[flat(min_index_x+1, min_index_y+1, min_index_z+1, grid_res_x, grid_res_y, grid_res_z)];
       
        // Linear interpolate along the y axis
        const float ty = (reached_point_y - (bounding_box_min_y + min_index_y * voxel_size)) / voxel_size;
        const float c0 = (1.f - ty) * c01 + ty * c23;
        const float c1 = (1.f - ty) * c45 + ty * c67;

        // Return final value interpolated along z
        const float tz = (reached_point_z - (bounding_box_min_z + min_index_z * voxel_size)) / voxel_size;      

        return (1.f - tz) * c0 + tz * c1;
    }

    // Compute the intersection of the ray and the grid
    // The intersection procedure uses ray marching to check if we have an interaction with the stored surface    
    __global__ void Intersect(
        const float* grid,
        const float* origins,
        const float* directions,
        const float* origin_image_distances, 
        const float* pixel_distances, 
        const float bounding_box_min_x,
        const float bounding_box_min_y,
        const float bounding_box_min_z,
        const float bounding_box_max_x,
        const float bounding_box_max_y,
        const float bounding_box_max_z,
        const int grid_res_x, 
        const int grid_res_y,
        const int grid_res_z,
        float* voxel_position,
        float* intersection_pos, 
        const int width, 
        const int height) {
        
        // Compute voxel size
        const float voxel_size = (bounding_box_max_x - bounding_box_min_x) / (grid_res_x - 1);

        // Define constant values
        const int max_steps = 1000;
        bool first_time = true;
        float depth = 0;
        int gotten_result = 0;

        const int pixel_index = blockIdx.x * blockDim.x + threadIdx.x;

        if (pixel_index < width * height) {

            const int i = 3 * pixel_index;
            
            for (int steps = 0; steps < max_steps; steps++) {

                float reached_point_x = origins[i] + depth * directions[i];
                float reached_point_y = origins[i+1] + depth * directions[i+1];
                float reached_point_z = origins[i+2] + depth * directions[i+2];

                // Get the signed distance value for the point the ray reaches
                const float distance = ValueAt(grid, reached_point_x, reached_point_y, reached_point_z,
                                               directions[i], directions[i+1], directions[i+2],
                                               bounding_box_min_x,
                                               bounding_box_min_y,
                                               bounding_box_min_z,
                                               bounding_box_max_x,
                                               bounding_box_max_y,
                                               bounding_box_max_z,
                                               grid_res_x, 
                                               grid_res_y,
                                               grid_res_z, first_time);
                first_time = false;

                // Check if the ray is going ourside the bounding box
                if (distance == -1) {
                    voxel_position[i] = -1;
                    voxel_position[i+1] = -1;
                    voxel_position[i+2] = -1;
                    intersection_pos[i] = -1;
                    intersection_pos[i+1] = -1;
                    intersection_pos[i+2] = -1;
                    gotten_result = 1;
                    break;
                }

                // Check if we are close enough to the surface
               if (distance < pixel_distances[pixel_index] / origin_image_distances[pixel_index] * depth && distance) {

                    // Compute the the minimum point of the intersecting voxel
                    voxel_position[i] = floorf((reached_point_x - bounding_box_min_x) / voxel_size);
                    voxel_position[i+1] = floorf((reached_point_y - bounding_box_min_y) / voxel_size);
                    voxel_position[i+2] = floorf((reached_point_z - bounding_box_min_z) / voxel_size);
                    if (voxel_position[i] == grid_res_x - 1) {
                        voxel_position[i] = voxel_position[i] - 1;
                    }
                    if (voxel_position[i+1] == grid_res_x - 1) {
                        voxel_position[i+1] = voxel_position[i+1] - 1;
                    }
                    if (voxel_position[i+2] == grid_res_x - 1) {
                        voxel_position[i+2] = voxel_position[i+2] - 1;
                    }
                    intersection_pos[i] = reached_point_x;
                    intersection_pos[i+1] = reached_point_y;
                    intersection_pos[i+2] = reached_point_z;
                    gotten_result = 1;
                    break;
                }

                // Increase distance
                depth += distance;

            }
            
            if (gotten_result == 0) {

                // No intersections
                voxel_position[i] = -1;
                voxel_position[i+1] = -1;
                voxel_position[i+2] = -1;
                intersection_pos[i] = -1;
                intersection_pos[i+1] = -1;
                intersection_pos[i+2] = -1;
            }
        }
    }
} // namespace

// Ray marching to get the first corner position of the voxel the ray intersects
std::vector<at::Tensor> ray_matching_cuda(
                           const at::Tensor w_h_3,
                           const at::Tensor w_h,
                           const at::Tensor grid, 
                           const int width, 
                           const int height,
                           const float bounding_box_min_x,
                           const float bounding_box_min_y,
                           const float bounding_box_min_z,
                           const float bounding_box_max_x,
                           const float bounding_box_max_y,
                           const float bounding_box_max_z,
                           const int grid_res_x, 
                           const int grid_res_y,
                           const int grid_res_z, 
                           const float eye_x,  
                           const float eye_y,  
                           const float eye_z) {

    const int thread = 512;

    at::Tensor origins = at::zeros_like(w_h_3);
    at::Tensor directions = at::zeros_like(w_h_3);
    at::Tensor origin_image_distances = at::zeros_like(w_h);
    at::Tensor pixel_distances = at::zeros_like(w_h);

        GenerateRay<<<(width * height + thread - 1) / thread, thread>>>(
                                     origins.data<float>(), 
                                     directions.data<float>(), 
                                     origin_image_distances.data<float>(),
                                     pixel_distances.data<float>(), 
                                     width, 
                                     height,
                                     eye_x, 
                                     eye_y, 
                                     eye_z);

    at::Tensor voxel_position = at::zeros_like(w_h_3);
    at::Tensor intersection_pos = at::zeros_like(w_h_3);

    Intersect<<<(width * height + thread - 1) / thread, thread>>>(
                                                    grid.data<float>(), 
                                                    origins.data<float>(), 
                                                    directions.data<float>(),
                                                    origin_image_distances.data<float>(),
                                                    pixel_distances.data<float>(), 
                                                    bounding_box_min_x,
                                                    bounding_box_min_y,
                                                    bounding_box_min_z,
                                                    bounding_box_max_x,
                                                    bounding_box_max_y,
                                                    bounding_box_max_z,
                                                    grid_res_x, 
                                                    grid_res_y,
                                                    grid_res_z, 
                                                    voxel_position.data<float>(), 
                                                    intersection_pos.data<float>(), 
                                                    width, 
                                                    height);        

    return {intersection_pos, voxel_position, directions};
}



